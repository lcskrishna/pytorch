
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SpatialDilatedMaxPooling.cu"
#else

#include <THCUNN/common.h>
#include <THCUNN/generic/pooling_shape.h>
#include <ATen/cuda/CUDAContext.h>
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <c10/util/Exception.h>

#include <tuple>

#include <THH/THH.h>

#include <ATen/miopen/miopen-wrapper.h>
#include <ATen/miopen/Descriptors.h>
#include <ATen/miopen/Types.h>
#include <ATen/miopen/Utils.h>

#include <ATen/TensorUtils.h>

#include <functional>
#include <iterator>
#include <sstream>
#include <algorithm>
#include <memory>
#include <mutex>
#include <stdint.h>
#include <unordered_map>

#include <iostream>

static inline void THNN_(SpatialDilatedMaxPooling_shapeCheck)(
                         THCState *state,
                         THCTensor *input, THCTensor *gradOutput, THCIndexTensor *indices,
                         int kH, int kW, int dH, int dW, int padH, int padW,
                         int dilationH, int dilationW, bool ceil_mode) {

  THArgCheck(kW > 0 && kH > 0, 5,
             "kernel size should be greater than zero, but got kH: %d kW: %d", kH, kW);
  THArgCheck(dW > 0 && dH > 0, 8,
             "stride should be greater than zero, but got dH: %d dW: %d", dH, dW);
  THArgCheck(dilationH > 0 && dilationW > 0, 12,
             "dilation should be greater than zero, but got dilationH: %d dilationW: %d",
             dilationH, dilationW);

  int ndim = input->dim();
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;
  int batchSize = 1;

  if (ndim == 4) {
    batchSize = input->size(0);
    dimf++;
    dimh++;
    dimw++;
  }

  THCUNN_argCheck(state, !input->is_empty() && (ndim == 3 || ndim == 4), 2, input,
                  "non-empty 3D or 4D input tensor expected but got: %s");
  THArgCheck(kW/2 >= padW && kH/2 >= padH, 2,
             "pad should be smaller than half of kernel size, but got "
             "padW = %d, padH = %d, kW = %d, kH = %d",
             padW, padH, kW, kH);

  int64_t nInputPlane = input->size(dimh-1);
  int64_t nInputRows = input->size(dimh);
  int64_t nInputCols = input->size(dimw);
  int64_t nOutputPlane = nInputPlane;

  int64_t nOutputRows = pooling_output_shape<int64_t>(nInputRows, kH, padH, dH, dilationH, ceil_mode);
  int64_t nOutputCols = pooling_output_shape<int64_t>(nInputCols, kW, padW, dW, dilationW, ceil_mode);

  if (nOutputCols < 1 || nOutputRows < 1)
    THError("Given input size: (%dx%dx%d). "
            "Calculated output size: (%dx%dx%d). Output size is too small",
            nInputPlane,nInputRows,nInputCols,nInputPlane,nOutputRows,nOutputCols);

  if (gradOutput != NULL) {
    THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimh, nOutputRows);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimw, nOutputCols);
  }
  if (indices != NULL) {
    THCUNN_check_dim_size_indices(state, indices, 4, 0, batchSize);
    THCUNN_check_dim_size_indices(state, indices, 4, 1, nOutputPlane);
    THCUNN_check_dim_size_indices(state, indices, 4, 2, nOutputRows);
    THCUNN_check_dim_size_indices(state, indices, 4, 3, nOutputCols);
  }
}

void THNN_(SpatialDilatedMaxPooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int dilationW, int dilationH,
           bool ceil_mode)
{

  THCUNN_assertSameGPU(state, 3, input, output, indices);
  THNN_(SpatialDilatedMaxPooling_shapeCheck)
       (state, input, NULL, NULL, kH, kW, dH, dW,
        padH, padW, dilationH, dilationW, ceil_mode);

  int64_t nInputCols, nInputRows, nInputPlane, batchSize;
  int64_t nOutputCols, nOutputRows;

  if (input->dim() == 3) {
    nInputCols = input->size(2);
    nInputRows = input->size(1);
    nInputPlane = input->size(0);
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size(3);
    nInputRows = input->size(2);
    nInputPlane = input->size(1);
    batchSize = input->size(0);
  }

  nOutputCols = pooling_output_shape<int64_t>(nInputCols, kW, padW, dW, dilationW, ceil_mode);
  nOutputRows = pooling_output_shape<int64_t>(nInputRows, kH, padH, dH, dilationH, ceil_mode);

  input = THCTensor_(newContiguous)(state, input);
  scalar_t* input_data = THCTensor_(data)(state, input);

  THCTensor_(resize4d)(state, output, batchSize, nInputPlane, nOutputRows, nOutputCols);
  THCUNN_resizeAs_indices(state, indices, output);

  THCIndex_t* indices_data = THCIndexTensor_(data)(state, indices);
  scalar_t* output_data = THCTensor_(data)(state, output);

  int count = THCTensor_(nElement)(state, output);
#if defined (__HIP_PLATFORM_HCC__)
  int kernel_size[2] = {kH, kW};
  int stride[2] = {dH, dW};
  int padding[2] = {padH, padW};
  int dilation[2] = {dilationH, dilationW};

  //Write an miopen implementation.
  miopenPoolingMode_t mode = miopenPoolingMax;
  auto handle = at::native::getMiopenHandle();
  miopenDataType_t datatype = miopenFloat;

  //Input and output tensor descriptors.
  miopenTensorDescriptor_t idesc;
  miopenTensorDescriptor_t odesc;
  miopenCreateTensorDescriptor(&idesc);
  miopenCreateTensorDescriptor(&odesc);

  miopenSet4dTensorDescriptor(idesc, datatype, batchSize, nInputPlane, nInputCols, nInputRows);
  miopenSet4dTensorDescriptor(odesc, datatype, batchSize, nInputPlane, nOutputCols, nOutputRows);

  //Pooling Descriptor.
  miopenPoolingDescriptor_t pdesc;
  miopenCreatePoolingDescriptor(&pdesc);
  miopenSet2dPoolingDescriptor(pdesc, mode, kH, kW, padH, padW, dH, dW);

  //Get workspace size.
  size_t ws_size;
  miopenPoolingGetWorkSpaceSize(odesc, &ws_size);

  at::native::Constant one(datatype, 1);
  at::native::Constant zero(datatype, 0);

  miopenPoolingForward(handle, pdesc, &one, idesc, (void *) input_data, &zero, odesc, (void *) output_data, true, indices_data, ws_size);
  indices_data = (THCIndex_t *) indices_data;


  //Destroy descriptors.
  miopenDestroyPoolingDescriptor(pdesc);
  miopenDestroyTensorDescriptor(odesc);
  miopenDestroyTensorDescriptor(idesc);

#else
  MaxPoolForward<scalar_t, accreal> <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, input_data,
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW, padH, padW, dilationH, dilationW, output_data, indices_data);
  THCudaCheck(cudaGetLastError());
#endif
  if(input->dim() == 3)
    THCTensor_(resize3d)(state, output, nInputPlane, nOutputRows, nOutputCols);



  THCTensor_(free)(state, input);
}

void THNN_(SpatialDilatedMaxPooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int dilationW, int dilationH,
           bool ceil_mode)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, indices, gradInput);
  THNN_(SpatialDilatedMaxPooling_shapeCheck)
       (state, input, gradOutput, indices, kH, kW, dH, dW,
       padH, padW, dilationH, dilationW, ceil_mode);

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  int64_t nInputCols, nInputRows, nInputPlane, batchSize;
  int64_t nOutputCols, nOutputRows;

  if (THTensor_nDimensionLegacyAll(input) == 3) {
    nInputCols = input->size(2);
    nInputRows = input->size(1);
    nInputPlane = input->size(0);
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size(3);
    nInputRows = input->size(2);
    nInputPlane = input->size(1);
    batchSize = input->size(0);
  }

  nOutputCols = pooling_output_shape<int64_t>(nInputCols, kW, padW, dW, dilationW, ceil_mode);
  nOutputRows = pooling_output_shape<int64_t>(nInputRows, kH, padH, dH, dilationH, ceil_mode);

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  int count = THCTensor_(nElement)(state, input);
  dim3 grid;
  int imgcount = nInputCols * nInputRows;
  const int blocks = (imgcount + BACKWARD_THREADS - 1) / BACKWARD_THREADS;
  grid.x = blocks;
  grid.y = batchSize;
  grid.z = nInputPlane;
  uint64_t maxGridY = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
  uint64_t maxGridZ = at::cuda::getCurrentDeviceProperties()->maxGridSize[2];
  if (maxGridY < grid.y) grid.y = maxGridY;
  if (maxGridZ < grid.z) grid.z = maxGridZ;

#if defined (__HIP_PLATFORM_HCC__)
  
  miopenPoolingMode_t mode = miopenPoolingMax;
  auto handle = at::native::getMiopenHandle();
  miopenDataType_t datatype = miopenFloat;

  scalar_t * input_data = THCTensor_(data)(state, input);
  scalar_t * gradOutput_data = THCTensor_(data)(state, gradOutput);
  scalar_t * indices_data = THCTensor_(data)(state, indices);
  scalar_t * grad_input_data = THCTensor_(data)(state, gradInput);
  scalar_t * output_data = THCTensor_(data)(state, input);

  //Create tensor descriptors.
  miopenTensorDescriptor_t yDesc, dyDesc, xDesc, dxDesc; 
  miopenCreateTensorDescriptor(&yDesc);
  miopenCreateTensorDescriptor(&dyDesc);
  miopenCreateTensorDescriptor(&xDesc);
  miopenCreateTensorDescriptor(&dxDesc);

  miopenSet4dTensorDescriptor(yDesc, datatype, batchSize, nInputPlane, nInputCols, nInputRows);
  miopenSet4dTensorDescriptor(dyDesc, datatype, batchSize, nInputPlane, nInputCols, nInputRows);
  miopenSet4dTensorDescriptor(xDesc, datatype, batchSize, nInputPlane, nOutputCols, nOutputRows);
  miopenSet4dTensorDescriptor(dxDesc, datatype, batchSize, nInputPlane, nOutputCols, nOutputRows);

  //Pooling descriptor.
  miopenPoolingDescriptor_t pdesc;
  miopenCreatePoolingDescriptor(&pdesc);
  miopenSet2dPoolingDescriptor(pdesc, mode, kH, kW, padH, padW, dH, dW);

  //Constants.
  at::native::Constant one(datatype, 1);
  at::native::Constant zero(datatype, 0);

  miopenPoolingBackward(handle, pdesc, &one, yDesc, (void *)input_data, dyDesc, (void *) gradOutput_data, xDesc, (void *) output_data, &zero, dxDesc, 
                      (void *) grad_input_data, (void *) indices_data );

#else  
  MaxPoolBackward<scalar_t, accreal> <<< grid, BACKWARD_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count,
      THCTensor_(data)(state, gradOutput),
      THCIndexTensor_(data)(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW, padH, padW, dilationH, dilationW,
      THCTensor_(data)(state, gradInput));
  THCudaCheck(cudaGetLastError());
#endif

  THCTensor_(free)(state, gradOutput);

  // clean
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif
